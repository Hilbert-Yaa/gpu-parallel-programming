#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512

#define wbCheck(stmt)                                                \
  do                                                                 \
  {                                                                  \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
      return -1;                                                     \
    }                                                                \
  } while (0)

__global__ void total(float *input, float *output, int len)
{
  extern __shared__ float sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  sdata[tid] = input[i] + input[i + blockDim.x];
  __syncthreads();
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0)
    output[blockIdx.x] = sdata[0];
}

int main(int argc, char **argv)
{
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = ceil(numInputElements / float(BLOCK_SIZE << 1));
  if (numInputElements % (BLOCK_SIZE << 1))
  {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  //Allocate GPU memory
  hipMalloc((void **)&deviceInput, numInputElements * sizeof(float));
  hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //Copy memory to the GPU
  hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //Initialize the grid and block dimensions

  wbTime_start(Compute, "Performing CUDA computation");
  //Launch the GPU Kernel
  dim3 dimGrid((numInputElements - 1) / BLOCK_SIZE + 1, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  total<<<dimGrid, dimBlock, (BLOCK_SIZE + 1) * sizeof(float)>>>(deviceInput, deviceOutput, numOutputElements);
  hipDeviceSynchronize();
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  for (ii = 1; ii < numOutputElements; ii++)
  {
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
