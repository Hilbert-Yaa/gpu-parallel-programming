#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
                                     int numAColumns, int numBRows,
                                     int numBColumns, int numCRows,
                                     int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  int Width = numAColumns;
  __shared__ float aTile[32][32], bTile[32][32]; // block dim set to 32.
  float entry = 0;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  for (int i = 0; i < Width / 32; ++i) {
    aTile[ty][tx] = A[y * numAColumns + i * 32 + tx];
    bTile[ty][tx] = B[(i * 32 + ty) * numAColumns + x];
    __syncthreads();
    for (int k = 0; k < 32; ++k) {
      entry += aTile[ty][k] * bTile[k][tx];
    }
    __syncthreads();
  }
  C[y * numCColumns + x] = entry;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =
      (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  wbCheck(hipMalloc(&deviceA, numARows * numAColumns * sizeof(float)));
  wbCheck(hipMalloc(&deviceB, numBRows * numBColumns * sizeof(float)));
  wbCheck(hipMalloc(&deviceC, numCRows * numCColumns * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  wbCheck(hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float),
                     hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlk(32, 32);
  dim3 dimGrid(numCColumns % dimBlk.x ? numCColumns / dimBlk.x + 1
                                      : numCColumns / dimBlk.x,
               numCRows % dimBlk.y ? numCRows / dimBlk.y + 1
                                   : numCRows / dimBlk.y);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<dimGrid, dimBlk>>>(deviceA, deviceB, deviceC, numARows,
                                            numAColumns, numBRows, numBColumns,
                                            numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  wbCheck(hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
